#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "dehaze_device.h"

__global__ void rgbminKernel(unsigned char* input, unsigned int* rgbmin, int rows, int cols, unsigned int* haze);

__global__ void darkChannelKernel(unsigned int* rgbmin, int rows, int cols);

__global__ void reduceKernel(unsigned int* rein, int n);

__global__ void estimateTransmissionKernel(unsigned int* darkChannel, unsigned int* transmission, unsigned int lightA, int rows, int cols);

__global__ void getDehazedKernel(unsigned int* haze, unsigned int* transmission, unsigned char* dehazed, unsigned int lightA, int rows, int cols);

void darkChannelOnDevice(unsigned char* input, unsigned int* rgbmin, int rows, int cols, unsigned int* haze)
{
	dim3 dimBlock(32, 32);
	dim3 dimGrid((cols - 1)/dimBlock.x + 1, (rows-1)/dimBlock.y +1);
	rgbminKernel<<<dimGrid, dimBlock>>>(input, rgbmin, rows, cols, haze);
	hipDeviceSynchronize();
	dim3 dimGrid2((cols-3)/30+1, (rows-3)/30+1);
	darkChannelKernel<<<dimGrid2, dimBlock>>>(rgbmin, rows, cols);
	hipDeviceSynchronize();
}

void estimateAOnDevice(unsigned int* rein, int size)
{
	int n = size;
	while (n>1) {
		reduceKernel<<<(n-1)/1024+1, 1024>>>(rein, n);
		n = (n-1)/1024+1;
	}
	hipDeviceSynchronize();
}

void estimateTransmissionOnDevice(unsigned int* darkChannel, unsigned int* transmission, unsigned int lightA, int rows, int cols)
{
	dim3 dimBlock(32, 32);
	dim3 dimGrid((cols - 1)/dimBlock.x + 1, (rows-1)/dimBlock.y +1);
	estimateTransmissionKernel<<<dimGrid, dimBlock>>>(darkChannel, transmission, lightA, rows, cols);
	hipDeviceSynchronize();
}

void getDehazedOnDevice(unsigned int* haze, unsigned int* transmission, unsigned char* dehazed, unsigned int lightA, int rows, int cols)
{
	dim3 dimBlock(32, 32);
	dim3 dimGrid((cols - 1)/dimBlock.x + 1, (rows-1)/dimBlock.y +1);
	getDehazedKernel<<<dimGrid, dimBlock>>>(haze, transmission, dehazed, lightA, rows, cols);
	hipDeviceSynchronize();
}

__global__ void rgbminKernel(unsigned char* input, unsigned int* rgbmin, int rows, int cols, unsigned int* haze)
{
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;
    int row = threadIdx.y;
	int col = threadIdx.x;

	unsigned char intensity0 = 0x00;
	unsigned char intensity1 = 0x00;
	unsigned char intensity2 = 0x00;
	unsigned char pixelmin = 0x00;

	if ((32 * blockRow + row) < rows && (32 * blockCol + col) < cols)
	{
		intensity0 = input[(blockRow * 32 + row) * cols * 3 + 3 * (blockCol * 32 + col) + 0];
		intensity1 = input[(blockRow * 32 + row) * cols * 3 + 3 * (blockCol * 32 + col) + 1];
		intensity2 = input[(blockRow * 32 + row) * cols * 3 + 3 * (blockCol * 32 + col) + 2];
//		rein get data
		unsigned char t0 = intensity0;
		unsigned char t1 = intensity1;
		unsigned char t2 = intensity2;
		haze[(blockRow * 32 + row) * cols * 3 + 3 * (blockCol * 32 + col) + 0] = (unsigned int)t0;
		haze[(blockRow * 32 + row) * cols * 3 + 3 * (blockCol * 32 + col) + 1] = (unsigned int)t1;
		haze[(blockRow * 32 + row) * cols * 3 + 3 * (blockCol * 32 + col) + 2] = (unsigned int)t2;
}
	else
	{
		intensity0 = 0;
		intensity1 = 0;
		intensity2 = 0;
	}
	syncthreads();
	unsigned char tmp = intensity1 < intensity2 ? intensity1 : intensity2;
	pixelmin = intensity0 < tmp ? intensity0 : tmp;

	if ((32 * blockRow + row) < rows && (32 * blockCol + col) < cols)
	{
		rgbmin[(blockRow * 32 + row) * cols + (blockCol * 32 + col)] = (unsigned int)pixelmin;
	}
}

__global__ void darkChannelKernel(unsigned int* rgbmin, int rows, int cols)
{
	__shared__ unsigned int sdata[32][32];
	unsigned int x=blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int y=blockIdx.y*blockDim.y+threadIdx.y;

	if ((x-2*blockIdx.x)<cols && (y-2*blockIdx.y)<rows) {
		sdata[threadIdx.y][threadIdx.x] = rgbmin[(y-2*blockIdx.y) * cols + (x-2*blockIdx.x)];
	}

	if (threadIdx.x>0 && threadIdx.x< 31 && threadIdx.y>0 && threadIdx.y<31) {
		if ((x-2*blockIdx.x)<(cols-1) && (y-2*blockIdx.y)<(rows-1)) {
			unsigned int window[9];
			window[0] = sdata[threadIdx.y-1][threadIdx.x-1];
			window[1] = sdata[threadIdx.y-1][threadIdx.x];
			window[2] = sdata[threadIdx.y-1][threadIdx.x+1];
			window[3] = sdata[threadIdx.y][threadIdx.x-1];
			window[4] = sdata[threadIdx.y][threadIdx.x];
			window[5] = sdata[threadIdx.y][threadIdx.x+1];
			window[6] = sdata[threadIdx.y+1][threadIdx.x-1];
			window[7] = sdata[threadIdx.y+1][threadIdx.x];
			window[8] = sdata[threadIdx.y+1][threadIdx.x+1];
			syncthreads();
		    // Order elements (only half of them)
		    for (unsigned int j=0; j<5; ++j)
		    {
		        // Find position of minimum element
		        unsigned int min=j;
		        for (unsigned int l=j+1; l<9; ++l)
		            if (window[l] < window[min])
		                min=l;

		        // Put found minimum element in its place
		        unsigned int temp=window[j];
		        window[j]=window[min];
		        window[min]=temp;

		        syncthreads();
		    }
		    rgbmin[(y-2*blockIdx.y) * cols + (x-2*blockIdx.x)] = window[4];
		}
	}
}

__global__ void reduceKernel(unsigned int* rein, int n)
{
	__shared__ unsigned int sdata[1024];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + tid;

	sdata[tid] = 0;

	if (i < n) {
		sdata[tid] = rein[i];
	}
	__syncthreads();

//	if (blockSize >= 1024) {
		if (tid < 512) {
			sdata[tid] = sdata[tid]>sdata[tid+512] ? sdata[tid]:sdata[tid+512];
		}
		__syncthreads();
//	}
//	if (blockSize >= 512) {
		if (tid < 256) {
			sdata[tid] = sdata[tid]>sdata[tid+256] ? sdata[tid]:sdata[tid+256];
		}
		__syncthreads();
//	}
//	if (blockSize >= 256) {
		if (tid < 128) {
			sdata[tid] = sdata[tid]>sdata[tid+128] ? sdata[tid]:sdata[tid+128];
		}
		__syncthreads();
//	}
//	if (blockSize >= 128) {
		if (tid < 64) {
			sdata[tid] = sdata[tid]>sdata[tid+64] ? sdata[tid]:sdata[tid+64];
		}
		__syncthreads();
//	}

	if (tid < 32) {
//		if (blockSize >= 64) {
			sdata[tid] = sdata[tid]>sdata[tid+32] ? sdata[tid]:sdata[tid+32];
//		}
//		if (blockSize >= 32) {
			sdata[tid] = sdata[tid]>sdata[tid+16] ? sdata[tid]:sdata[tid+16];
//		}
//		if (blockSize >= 16) {
			sdata[tid] = sdata[tid]>sdata[tid+8] ? sdata[tid]:sdata[tid+8];
//		}
//		if (blockSize >= 8) {
			sdata[tid] = sdata[tid]>sdata[tid+4] ? sdata[tid]:sdata[tid+4];
//		}
//		if (blockSize >= 4) {
			sdata[tid] = sdata[tid]>sdata[tid+2] ? sdata[tid]:sdata[tid+2];
//		}
//		if (blockSize >= 2) {
			sdata[tid] = sdata[tid]>sdata[tid+1] ? sdata[tid]:sdata[tid+1];
//		}
	}

	if (tid == 0) {
		rein[blockIdx.x] = sdata[0];
		__syncthreads();
	}
}

__global__ void estimateTransmissionKernel(unsigned int* darkChannel, unsigned int* transmission, unsigned int lightA, int rows, int cols)
{
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;
	int row = threadIdx.y;
	int col = threadIdx.x;

	unsigned int dark, trans;

	if ((32 * blockRow + row) < rows && (32 * blockCol + col) < cols){
		dark = darkChannel[(blockRow*32 + row)*cols + blockCol*32 + col];
		trans = (1 - 0.75 * dark/ lightA) * 255;
		transmission[(blockRow*32 + row)*cols + blockCol*32 + col] = trans;
	}
	syncthreads();
}

__global__ void getDehazedKernel(unsigned int* haze, unsigned int* transmission, unsigned char* dehazed, unsigned int lightA, int rows, int cols)
{
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;
	int row = threadIdx.y;
	int col = threadIdx.x;

	int haze0, haze1, haze2, trans, dehaze0, dehaze1, dehaze2;
	float tmin = 0.1;
	float tmax, td;
	unsigned char pixel0, pixel1, pixel2;

	if ((32 * blockRow + row) < rows && (32 * blockCol + col) < cols){
		haze0 = haze[(blockRow * 32 + row) * cols * 3 + 3 * (blockCol * 32 + col) + 0];
		haze1 = haze[(blockRow * 32 + row) * cols * 3 + 3 * (blockCol * 32 + col) + 1];
		haze2 = haze[(blockRow * 32 + row) * cols * 3 + 3 * (blockCol * 32 + col) + 2];
		trans = transmission[(blockRow*32 + row)*cols + blockCol*32 + col];
		td = (float) trans;
		tmax = (td/255) < tmin ? tmin : (td/255);
		dehaze0 = abs(((int)haze0 - (int)lightA) / tmax + (int)lightA) > 255 ? 255 : abs(((int)haze0 - (int)lightA) / tmax + (int)lightA);
		dehaze1 = abs(((int)haze1 - (int)lightA) / tmax + (int)lightA) > 255 ? 255 : abs(((int)haze1 - (int)lightA) / tmax + (int)lightA);
		dehaze2 = abs(((int)haze2 - (int)lightA) / tmax + (int)lightA) > 255 ? 255 : abs(((int)haze2 - (int)lightA) / tmax + (int)lightA);
		pixel0 = (unsigned char)dehaze0;
		pixel1 = (unsigned char)dehaze1;
		pixel2 = (unsigned char)dehaze2;
		dehazed[(blockRow * 32 + row) * cols * 3 + 3 * (blockCol * 32 + col) + 0] = pixel0;
		dehazed[(blockRow * 32 + row) * cols * 3 + 3 * (blockCol * 32 + col) + 1] = pixel1;
		dehazed[(blockRow * 32 + row) * cols * 3 + 3 * (blockCol * 32 + col) + 2] = pixel2;
	}
	syncthreads();
}

void* AllocateDeviceMemory(size_t size)
{
	void* deviceMemory;
	hipMalloc((void**)&deviceMemory, size);
	return deviceMemory;
}

void CopyToDevice(void* host, void* device, size_t size)
{
	hipMemcpy(device, host, size, hipMemcpyHostToDevice);
}

void CopyFromDevice(void* host, void* device, size_t size)
{
	hipMemcpy(host, device, size, hipMemcpyDeviceToHost);
}

void DeviceToDevice(void* dst, void*src, size_t size)
{
	hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);
}

void FreeDeviceMemory(void* deviceMemory)
{
	hipFree(deviceMemory);
}
